#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

// 2D float texture
texture<uchar4, 2, hipReadModeElementType> texRef;

// Simple scaling kernel
__global__ void scaleKernel(uchar4* output, int width, int height, int scale, int newW, int newH)
{
    // Calculate normalized texture coordinates
    unsigned int x = threadIdx.x * scale;
    unsigned int y = blockIdx.y * scale;

    float4 result = { 0.0, 0.0, 0.0, 0.0 };
    uchar4 tmp;

    for (unsigned int i=0;i<scale;i++) {
        for (unsigned int j=0;j<scale;j++) {
  
           tmp = tex2D(texRef, x+j, y+i);

           result.x += tmp.x;
           result.y += tmp.y;
           result.z += tmp.z;
           result.w += tmp.w;
        }
    }

    float sqr = scale*scale;

    tmp.x = result.x / sqr;
    tmp.y = result.y / sqr;
    tmp.z = result.z / sqr;
    tmp.w = result.w / sqr;
  
    output[blockIdx.y*newW + threadIdx.x] = tmp;      
}

// Simple scaling kernel
__global__ void scaleKernel16(uchar4* output, int width, int height)
{
   // TODO
}

extern "C" {

int cudaScale(unsigned char* in, int w, int h, int scale, unsigned char *out) 
{ 
    // Describe the texture as 4 unsigned bytes per element
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

    // Copy the input data into the device
    hipArray* inArray;
    hipMallocArray(&inArray, &channelDesc, w, h);
    hipMemcpyToArray(inArray, 0, 0, in, w*h*4, hipMemcpyHostToDevice);

    // Set texture parameters (clamp the coordinates, return single points, and do not normalize)
    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode     = hipFilterModePoint;
    texRef.normalized     = false;

    // Bind the array to the texture
    hipBindTextureToArray(texRef, inArray, channelDesc);

    int newW = w/scale;
    int newH = h/scale;

    // Allocate result output array in device memory
    uchar4* output;
    hipMalloc((void**)&output, newW * newH * 4);

    // Invoke kernel
    dim3 dimBlock(newW, 1);
    dim3 dimGrid(1, newH);

    scaleKernel<<<dimGrid, dimBlock>>>(output, w, h, scale, newW, newH);

    hipMemcpy(out, output, newW*newH*4, hipMemcpyDeviceToHost); 

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

    // Free device memory
    hipFreeArray(inArray);
    hipFree(output);

    return 0;
}

}
